
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <ctime>

#define TILE_WIDTH 32

__global__ void matrixMult(int *a, int *b, int *c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int sum = 0;
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main()
{
    int n;
    n=4;

    // allocate memory for matrices on host
    int *a = new int[n * n];
    int *b = new int[n * n];
    int *c = new int[n * n];

    // initialize matrices with random values
    std::srand(std::time(0));
    for (int i = 0; i < n * n; ++i) {
        a[i] = std::rand() % 10;
        b[i] = std::rand() % 10;
    }

    // allocate memory for matrices on device
    int *dev_a, *dev_b, *dev_c;
    hipMalloc(&dev_a, n * n * sizeof(int));
    hipMalloc(&dev_b, n * n * sizeof(int));
    hipMalloc(&dev_c, n * n * sizeof(int));

    // copy matrices from host to device
    hipMemcpy(dev_a, a, n * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * n * sizeof(int), hipMemcpyHostToDevice);

    // launch kernel
    dim3 dimGrid((n - 1) / TILE_WIDTH + 1, (n - 1) / TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    matrixMult<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, n);

    // copy result matrix from device to host
    hipMemcpy(c, dev_c, n * n * sizeof(int), hipMemcpyDeviceToHost);

    // print result matrix
 std::cout << "Result matrix:\n";
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << a[i * n + j] << " ";
        }
        std::cout << "\n";
    }
 std::cout << "Result matrix:\n";
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << b[i * n + j] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "Result matrix:\n";
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << c[i * n + j] << " ";
        }
        std::cout << "\n";
    }

    // free memory on device
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // free memory on host
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}

